/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//#include <helper_functions.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 4900001;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    
    // Multi-GPU
    int numDevs = 0;
    hipGetDeviceCount(&numDevs);

    if (numDevs == 0)
    {
	fprintf(stderr, "There is no GPU device!");
        exit(EXIT_FAILURE);
    }
   
    // step size between device
    size_t step = numElements/numDevs*sizeof(float); // not size/numDevs
    
    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate arry of pointers to store the device input vectors
    float **d_As = (float **)malloc(numDevs);
    
    if(d_As == NULL)
    {
	fprintf(stderr, "Failed to allocate array d_As to store pointers of device vectors!\n");
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < numDevs; ++i)
    {
	d_As[i] = NULL;
    }

    for(int i = 0; i < numDevs; ++i)
    {
        hipSetDevice(i);
        if (i == numDevs-1)
        {
    	   err = hipMalloc((void **)&d_As[i], size-i*step);
	   //printf("The size is %d bytes and step is %d bytes.\n", size, step);
	   //printf("The last device should allocate %d bytes.\n", size-i*step);
        }
	else
	{
    	   err = hipMalloc((void **)&d_As[i], size/numDevs);
        }
    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector A's part %d (error code %s)!\n", hipGetErrorString(err), i);
        	exit(EXIT_FAILURE);
    	}
	
    }
    // Allocate arry of pointers to store the device input vectors
    float **d_Bs = (float **)malloc(numDevs);
  
    if(d_Bs == NULL)
    {
	fprintf(stderr, "Failed to allocate array d_Bs to store pointers of device vectors!\n");
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < numDevs; ++i)
    {
	d_Bs[i] = NULL;
    }

    for(int i = 0; i < numDevs; ++i)
    {
        hipSetDevice(i);
        if (i == numDevs-1)
        {
    	   err = hipMalloc((void **)&d_Bs[i], size-i*step);
        }
	else
	{
    	   err = hipMalloc((void **)&d_Bs[i], size/numDevs);
        }
	
    	if (err != hipSuccess)
    	{
           fprintf(stderr, "Failed to allocate device vector B's part %d (error code %s)!\n", hipGetErrorString(err), i);
           exit(EXIT_FAILURE);
    	}

    }
    // Allocate arry of pointers to store the device input vectors
    float **d_Cs = (float **)malloc(numDevs);
    if(d_Cs == NULL)
    {
	fprintf(stderr, "Failed to allocate array d_Cs to store pointers of device vectors!\n");
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < numDevs; ++i)
    {
	d_Cs[i] = NULL;
    }

    for(int i = 0; i < numDevs; ++i)
    {
	//set device is very important
        hipSetDevice(i);
        if (i == numDevs-1)
        {
    	   err = hipMalloc((void **)&d_Cs[i], size-i*step);
        }
	else
	{
    	   err = hipMalloc((void **)&d_Cs[i], size/numDevs);
        }

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector C's part %d (error code %s)!\n", hipGetErrorString(err), i);
        	exit(EXIT_FAILURE);
    	}
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory

    printf("Copy input data from the host memory to the CUDA device\n");
    for(int i = 0; i < numDevs; ++i)
    {
	//set device is very important
        hipSetDevice(i);
        if (i == numDevs-1)
        {
    		err = hipMemcpy(d_As[i], h_A+(i*numElements/numDevs), size-i*step, hipMemcpyHostToDevice);
        }
	else
	{
    		err = hipMemcpy(d_As[i], h_A+(i*numElements/numDevs), step, hipMemcpyHostToDevice);
        }

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector A's part %d from host to device (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    }
    for(int i = 0; i < numDevs; ++i)
    {
	//set device is very important
        hipSetDevice(i);
        if (i == numDevs-1)
        {
    		err = hipMemcpy(d_Bs[i], h_B+(i*numElements/numDevs), size-i*step, hipMemcpyHostToDevice);
	}
	else
	{
    		err = hipMemcpy(d_Bs[i], h_B+(i*numElements/numDevs), step, hipMemcpyHostToDevice);
	}

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector B's part %d from host to device (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    }

    // cudaEvent_t is used to caculate the running time of the program on the devices
    hipEvent_t start;
    err = hipEventCreate(&start);

    if (err != hipSuccess)
    {
	fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_SUCCESS);
    }

    hipEvent_t stop;
    err = hipEventCreate(&stop);

    if (err != hipSuccess)
    {
	fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_SUCCESS);
    }

    // Record the start event
    err = hipEventRecord(start, NULL);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements/numDevs + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < numDevs; ++i)
    {
    	printf("CUDA kernel %d launch with %d blocks of %d threads\n", i, blocksPerGrid, threadsPerBlock);
        hipSetDevice(i);
	if (i == numDevs-1)
	{
    		vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_As[i], d_Bs[i], d_Cs[i], numElements - i*numElements/numDevs);
	}
	else
	{
    		vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_As[i], d_Bs[i], d_Cs[i], numElements/numDevs);
	}
    	err = hipGetLastError();

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to launch vectorAdd kernel %d (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    }

    // Record the stop event
    err = hipEventRecord(stop, NULL);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    err = hipEventSynchronize(stop);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("The total time is %f msec.\n", msecTotal);
    
    for (int i = 0; i < numDevs; ++i)
    {
	hipSetDevice(i);
	if (i == numDevs-1)
	{
    		err = hipMemcpy(h_C+(i*numElements/numDevs), d_Cs[i], size-i*step, hipMemcpyDeviceToHost);
	}
	else
	{
    		err = hipMemcpy(h_C+(i*numElements/numDevs), d_Cs[i], step, hipMemcpyDeviceToHost);
	}

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector C's part %d from device to host (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    }
    
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free device global memory
    for(int i = 0; i < numDevs; ++i)
    {
    	err = hipFree(d_As[i]);

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to free device vector A's part %d (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    }
    for(int i = 0; i < numDevs; ++i)
    {
    	err = hipFree(d_Bs[i]);

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to free device vector B's part %d (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    }
    for(int i = 0; i < numDevs; ++i)
    {
    	err = hipFree(d_Cs[i]);

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to free device vector C's part %d (error code %s)!\n", i, hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    }
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

